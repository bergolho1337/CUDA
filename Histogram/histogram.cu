// Versao serial do problema do histograma na CPU
// Tempo = 5.811268e+02 ms

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "book.h"

const int SIZE = (100*1024*1024);

int main ()
{
    // Varaiveis de medicao de tempo
    hipEvent_t start, stop;
    int i;

    // Vetor randomico de 'char' de tamanho SIZE
    unsigned char *buffer = (unsigned char*)big_random_block(SIZE);
    unsigned int  histo[256];

    // Inicializa o contador do histograma com 0's
    for (i = 0; i < 256; i++)
      histo[i] = 0;

    // Inicializa as variaveis de medicao
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start,0));

    // Percorre os dados do buffer contando quantas vezes
    // cada elemento apareceu
    for (i = 0; i < SIZE; i++)
      histo[buffer[i]]++;

    // Para o cronometro e calcula o tempo
    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));

    // Confere se a conta esta correta e imprime o tempo em milisegundos
    long histoCount = 0;
    for (i = 0; i < 256; i++)
      histoCount += histo[i];
    printf("Histogram Sum: %ld\n",histoCount);
    printf("This result must the same as SIZE: %ld\n",SIZE);
    printf("Time to generate: %e ms\n",elapsedTime);
    return 0;
}
