
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

const int N = 8;
const int BLOCKSIZE = 8;
const int GRIDSIZE = 1;

// ---------------------------------------------- KERNELS ---------------------------------------------------------------
// Hillis Steele Scan - Inclusive Scan
__global__ void gpu_blelloch_scan (int *in, int *out)
{
    extern __shared__ int cache[];
    int myId = threadIdx.x;
    // Copy the array to shared memory
    cache[myId] = in[myId];
    __syncthreads();

    for (int d = 1; d < N; d <<= 1)
    {
        if (myId >= d)
            cache[myId] += cache[myId - d];
        __syncthreads();
    }

    // Write results to output
    // -- Inclusive
    //out[myId] = cache[myId];       

    // -- Exclusive
    if (myId == 0) out[0] = 0;
    else out[myId] = cache[myId-1];

}
// ---------------------------------------------------------------------------------------------------------------------------
// -------------------------------------------------- CPU Functions ----------------------------------------------------------

void print (int *v)
{
    for (int i = 0; i < N; i++)
        printf("%d ",v[i]);
    printf("\n\n");
}

void generate (int *v)
{
    for (int i = 0; i < N; i++)
        v[i] = i+1;
    /*
    v[0] = 13;
    v[1] = 7;
    v[2] = 16;
    v[3] = 21;
    v[4] = 8;
    v[5] = 20;
    v[6] = 13;
    v[7] = 12;
    */
} 

void Usage (char pName[])
{
    printf("============================================\n");
    printf("Usage:> %s \n",pName);
    printf("============================================\n");
}

// ---------------------------------------------------------------------------------------------------------------------------
// ------------------------------------------------ MAIN FUNCTION ------------------------------------------------------------

int main (int argc, char *argv[])
{
    if (argc-1 != 0)
    {
        Usage(argv[0]);
        exit(1);
    }

    // Declare and allocate memory for the host and device structures
    int *h_in, *h_out;
    int *d_in, *d_out;
    size_t sizeIn = N*sizeof(int);
    size_t sizeOut = N*sizeof(int);
    h_in = (int*)malloc(sizeIn); generate(h_in); print(h_in);
    hipMalloc(&d_in,sizeIn);
    hipMemcpy(d_in,h_in,sizeIn,hipMemcpyHostToDevice);

    h_out = (int*)malloc(sizeOut);
    hipMalloc(&d_out,sizeOut);

    dim3 gridSize(1,1);
    dim3 blockSize(BLOCKSIZE,1);
    size_t sharedMem = sizeof(int)*BLOCKSIZE*2;

    // Call reduce kernel
    gpu_blelloch_scan<<<gridSize,blockSize,sharedMem>>>(d_in,d_out);
    hipMemcpy(h_out,d_out,sizeOut,hipMemcpyDeviceToHost);

    // Print the result
    print(h_out);

    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);

    return 0;
}