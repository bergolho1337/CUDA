
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

const int N = 8;
const int BLOCKSIZE = 8;
const int GRIDSIZE = 1;

// ---------------------------------------------- KERNELS ---------------------------------------------------------------
__global__ void gpu_inclusive_scan (int *in, int *out)
{
    extern __shared__ int cache[];
    int tid = threadIdx.x;
    int offset = 1;
    // Load the input into shared memory
    cache[2*tid] = in[2*tid]; 
    cache[2*tid+1] = in[2*tid+1];
    __syncthreads();

    // Build sum in place up the tree
    for (int d = N >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (tid < d)
        {
            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;

            cache[bi] += cache[ai];
        }
        offset *= 2;
    }

    // Clear the last element
    if (tid == 0)
    {
        cache[N-1] = 0;
    }

    // Transverse down and build scan
    for (int d = 1; d < N; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (tid < d)
        {
            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;

            int aux = cache[ai];
            cache[ai] = cache[bi];
            cache[bi] += aux;
        }
    }
    __syncthreads();

    // Write results to output
    out[2*tid] = cache[2*tid];
    out[2*tid+1] = cache[2*tid+1];
}
// ---------------------------------------------------------------------------------------------------------------------------
// -------------------------------------------------- CPU Functions ----------------------------------------------------------

void print (int *v)
{
    for (int i = 0; i < N; i++)
        printf("%d ",v[i]);
    printf("\n\n");
}

void generate (int *v)
{
    for (int i = 0; i < N; i++)
        v[i] = i+1;
    /*
    v[0] = 13;
    v[1] = 7;
    v[2] = 16;
    v[3] = 21;
    v[4] = 8;
    v[5] = 20;
    v[6] = 13;
    v[7] = 12;
    */
} 

void Usage (char pName[])
{
    printf("============================================\n");
    printf("Usage:> %s \n",pName);
    printf("============================================\n");
}

// ---------------------------------------------------------------------------------------------------------------------------
// ------------------------------------------------ MAIN FUNCTION ------------------------------------------------------------

int main (int argc, char *argv[])
{
    if (argc-1 != 0)
    {
        Usage(argv[0]);
        exit(1);
    }

    // Declare and allocate memory for the host and device structures
    int *h_in, *h_out;
    int *d_in, *d_out;
    size_t sizeIn = N*sizeof(int);
    size_t sizeOut = N*sizeof(int);
    h_in = (int*)malloc(sizeIn); generate(h_in); print(h_in);
    hipMalloc(&d_in,sizeIn);
    hipMemcpy(d_in,h_in,sizeIn,hipMemcpyHostToDevice);

    h_out = (int*)malloc(sizeOut);
    hipMalloc(&d_out,sizeOut);

    dim3 gridSize(1,1);
    dim3 blockSize(BLOCKSIZE,1);
    size_t sharedMem = sizeof(int)*BLOCKSIZE*2;

    // Call reduce kernel
    gpu_inclusive_scan<<<gridSize,blockSize,sharedMem>>>(d_in,d_out);
    hipMemcpy(h_out,d_out,sizeOut,hipMemcpyDeviceToHost);

    // Print the result
    print(h_out);

    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);

    return 0;
}